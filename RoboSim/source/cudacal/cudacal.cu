#include "hip/hip_runtime.h"
#include "cudacal.h"

__global__ void calculateVerticesAndTriangles(__pointkernel__* d_vertices, __vertikernel__* d_triangles, float* d_data,
    size_t numRows, size_t numCols, float x_spacing, float y_spacing) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int idx = row * numCols + col;

        // Calculate vertex position
        d_vertices[idx].x = col * x_spacing;
        d_vertices[idx].y = row * y_spacing;
        d_vertices[idx].z = d_data[idx];

        // Calculate triangles
        if (row < numRows - 1 && col < numCols - 1) {
            int top_left = idx;
            int top_right = idx + 1;
            int bottom_left = idx + numCols;
            int bottom_right = bottom_left + 1;

            d_triangles[2 * idx].v1 = top_left;
            d_triangles[2 * idx].v2 = top_right;
            d_triangles[2 * idx].v3 = bottom_left;

            d_triangles[2 * idx + 1].v1 = top_right;
            d_triangles[2 * idx + 1].v2 = bottom_right;
            d_triangles[2 * idx + 1].v3 = bottom_left;
        }
    }
}

extern "C" void launchKernel(float* d_data, __pointkernel__ * d_vertices, __vertikernel__ * d_triangles,
    size_t numRows, size_t numCols, float x_spacing, float y_spacing) {
    dim3 blockSize(16, 16);
    dim3 gridSize((numCols + blockSize.x - 1) / blockSize.x, (numRows + blockSize.y - 1) / blockSize.y);

    calculateVerticesAndTriangles << <gridSize, blockSize >> > (d_vertices, d_triangles, d_data, numRows, numCols, x_spacing, y_spacing);

    hipDeviceSynchronize();
}
